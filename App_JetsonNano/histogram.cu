#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <opencv4/opencv2/core/hal/interface.h>
#include <opencv4/opencv2/core/matx.hpp>
#include <opencv4/opencv2/highgui.hpp>
#include <opencv4/opencv2/imgcodecs.hpp>
#include <opencv4/opencv2/opencv.hpp>
#include <opencv4/opencv2/imgproc.hpp>
#include <opencv4/opencv2/core/core.hpp>
#include <chrono>

struct IN_TYPE {
    char r;
    char g;
    char b;
};

#define NUM_BINS 256
#define NUM_PARTS 256*3

#define NUM_SUB_HIST_1 5
#define NUM_SUB_HIST_2 50

__global__ void histogram_gmem_atomics(const IN_TYPE *in, int width, int height, unsigned int *out)
{

   //printf("%d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
   // pixel coordinates
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // grid dimensions
  int nx = blockDim.x * gridDim.x; 
  int ny = blockDim.y * gridDim.y;

  // linear thread index within 2D block
  int t = threadIdx.x + threadIdx.y * blockDim.x; 

  // total threads in 2D block
  int nt = blockDim.x * blockDim.y; 
  
  // linear block index within 2D grid
  int g = blockIdx.x + blockIdx.y * gridDim.x;

  // initialize temporary accumulation array in global memory
  unsigned int *gmem = out + g * NUM_PARTS;
  for (int i = t; i < 3 * NUM_BINS; i += nt) gmem[i] = 0;

 // printf("x: %d y: %d nx:%d ny:%d t:%d nt:%d g:%d w:%d h:%d\n", x,y,nx,ny,t,nt,g,width,height);

  // process pixels
  // updates our block's partial histogram in global memory
  for (int col = x; col < width; col += nx) 
    for (int row = y; row < height; row += ny) { 
      char r = (in[row * width + col].r);
      char g = (in[row * width + col].g);
      char b = (in[row * width + col].b);
      //printf("r:%d g:%d b:%d\n", r, g, b);
      atomicAdd(&gmem[NUM_BINS * 0 + r], 1);
      atomicAdd(&gmem[NUM_BINS * 1 + g], 1);
      atomicAdd(&gmem[NUM_BINS * 2 + b], 1);
    }
}

__global__ void histogram_final_accum(const unsigned int *in, int n, unsigned int *out)
//__global__ void histogram_final_accum()
{
  //printf("Something");
  //printf("%d %d %d\n", blockIdx.x , blockDim.x , threadIdx.x);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < 3 * NUM_BINS) {
    unsigned int total = 0;
    for (int j = 0; j < n; j++) 
      total += in[i + NUM_PARTS * j];
    
    out[i] = total;
    //out[i] = 57;
  }else
    out[i] = 101;
  
}

unsigned int* histogram_gpu(int num_threads_x, int num_threads_y, IN_TYPE *pixel_array, int cols, int rows){
    unsigned int *out, *out2;

    hipMallocManaged(&out,  sizeof(unsigned int)*3*NUM_BINS*num_threads_x*num_threads_y);
    hipMallocManaged(&out2, sizeof(unsigned int)*3*NUM_BINS);


    histogram_gmem_atomics<<<num_threads_x,num_threads_y>>>(pixel_array, cols, rows, out);
    hipDeviceSynchronize();

    histogram_final_accum<<<3,255>>>(out, num_threads_x*num_threads_y, out2);
    hipDeviceSynchronize();
    hipFree(out);
    return out2;
}

void histogram_cpu(IN_TYPE *pixel_array, int cols, int rows, int* hist){

    for (int i = 0; i < cols; i++)
    {
        for(int j = 0; j<rows;j++)
        {
            char r = ( pixel_array[j * cols + i].r);
            char g = ( pixel_array[j * cols + i].g);
            char b = ( pixel_array[j * cols + i].b);
	    hist[NUM_BINS * 0 + r]++;
	    hist[NUM_BINS * 1 + g]++;
	    hist[NUM_BINS * 2 + b]++;
        }
    }

}

int main(void)
{
    // File to write the Histogram into
    std::ofstream histFile ("histogram.txt");
    cv::Mat inputFile = cv::imread("hist_rainbow.png", cv::IMREAD_COLOR);

    IN_TYPE *pixel_array;

    IN_TYPE *pixel_array_cpu = (IN_TYPE*)malloc(sizeof(IN_TYPE)*inputFile.cols*inputFile.rows);

    hipMallocManaged(&pixel_array,  sizeof(IN_TYPE)*inputFile.cols*inputFile.rows);

    

    std::cout << "BEGINING COLOR CHANNELS ..." << std::endl;
    // Variable to store the byte from each channel
    cv::Vec3b byteFromPixel;

    int pixel_idx = 0;


    // Get all the bytes from the red channel 

    for (int i = 0; i < inputFile.rows; i++)
    {
        for(int j = 0; j<inputFile.cols;j++)
        {
            // Store the byte in each corresponding file according to channel
            byteFromPixel = inputFile.at<cv::Vec3b>(i,j);
            // byteArray[byteFromPixel]++;
    
            pixel_array[pixel_idx].b = byteFromPixel[0];
            pixel_array[pixel_idx].g = byteFromPixel[1];
            pixel_array[pixel_idx].r = byteFromPixel[2];

            pixel_array_cpu[pixel_idx].b = byteFromPixel[0];
            pixel_array_cpu[pixel_idx].g = byteFromPixel[1];
            pixel_array_cpu[pixel_idx].r = byteFromPixel[2];
            pixel_idx++;
        }
    }

    std::chrono::steady_clock::time_point begin, end;

    int n_threads[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};
    unsigned int* hist;


    for(int i = 0; i < 9; i++){
	for (int j = i; j < 9; j++){
	       begin = std::chrono::steady_clock::now();

	       hist = histogram_gpu(n_threads[j], n_threads[i], pixel_array, inputFile.cols, inputFile.rows);

	       end = std::chrono::steady_clock::now();

	       printf("Done run GPU (%3dx%3d threads): %8d us\n",n_threads[j], n_threads[i], std::chrono::duration_cast<std::chrono::microseconds>(end-begin));
		hipFree(hist);
	}
    }

    int hist_cpu[NUM_BINS*3] = {};
    begin = std::chrono::steady_clock::now();

    histogram_cpu( pixel_array_cpu, inputFile.cols, inputFile.rows, hist_cpu);

    end = std::chrono::steady_clock::now();

    printf("Done run cpu:\nTime: %d ms\n\n", std::chrono::duration_cast<std::chrono::microseconds>(end-begin));

    for(int k = 0; k < 3;k++)
    {
        for(int i = 0; i < NUM_BINS; i++){
            // Print
           histFile << hist_cpu[k*NUM_BINS+i] << std::endl;
        }
    }

    hipFree(pixel_array);
}
